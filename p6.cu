
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
#define N 16
#define BLOCK_DIM 16

using namespace std;

__global__ void multiply(float A[], float B[], float C[])
{
        __shared__ float sub_A[BLOCK_DIM][BLOCK_DIM], sub_B[BLOCK_DIM][BLOCK_DIM];
        int global_x = threadIdx.x + blockIdx.x * blockDim.x, global_y = threadIdx.y + blockIdx.y * blockDim.y, global_ID = global_y * N + global_x;
        C[global_ID] = 0;
        for(int i = 0; i < N / BLOCK_DIM; i++)
        {
         	sub_A[threadIdx.y][threadIdx.x] = A[global_y * N + global_x + BLOCK_DIM * i];
         	sub_B[threadIdx.y][threadIdx.x] = B[(global_y + BLOCK_DIM * i) * N + global_x];	       
        	__syncthreads();
        	for(int j = 0; j < BLOCK_DIM; j++)
        		C[global_ID] += sub_A[threadIdx.y][j] * sub_B[j][threadIdx.x];
		__syncthreads();
        }
}

void init_matrix(float mat[])
{
	for(int i = 0; i < N; i++)
		for(int j = 0; j < N; j++)
			mat[i * N + j] = 1;
}

void print_matrix(float mat[])
{

	for(int i = 0; i < N; i++)
	{
		for(int j = 0; j < N; j++)
			cout << mat[i * N + j] << "  ";
		cout << endl;
	}
	cout << endl;
}

int main()
{
	float *A = new float[N * N], *B = new float[N * N], *C = new float[N * N], *cuda_A, *cuda_B, * cuda_C;
	init_matrix(A);
	cout << "A : " << endl;
	print_matrix(A);
	init_matrix(B);
	cout << "B : " << endl;
	print_matrix(B);
	hipMalloc(&cuda_A, sizeof(float) * N * N);
	hipMalloc(&cuda_B, sizeof(float) * N * N);
	hipMalloc(&cuda_C, sizeof(float) * N * N);
	hipMemcpy(cuda_A, A, sizeof(float) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(cuda_B, B, sizeof(float) * N * N, hipMemcpyHostToDevice);
	dim3 grid_dim(N / BLOCK_DIM, N / BLOCK_DIM), block_dim(BLOCK_DIM, BLOCK_DIM);
	multiply<<<grid_dim, block_dim>>>(cuda_A, cuda_B, cuda_C);
	hipMemcpy(C, cuda_C, sizeof(float) * N * N, hipMemcpyDeviceToHost);
	cout << "C : " << endl;
	print_matrix(C);	
	return 0;
}
