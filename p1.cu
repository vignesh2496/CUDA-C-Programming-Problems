#include "hip/hip_runtime.h"
#include<stdio.h>

int main() 
{
	int n_devices;
	hipGetDeviceCount(&n_devices);
  	for (int i = 0; i < n_devices; i++)
  	{
    		hipDeviceProp_t prop;
    		hipGetDeviceProperties(&prop, i);
    		printf("  Device number: %d\n", i);
    		printf("  Device name: %s\n", prop.name);
    		printf("  Memory clock rate (KHz): %d\n", prop.memoryClockRate);
    		printf("  Memory bus width (bits): %d\n", prop.memoryBusWidth);
    		printf("  Peak memory bandwidth (GB/s): %f\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
		printf("  Maximum number of grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("  Total constant memory: %d\n", prop.totalConstMem);
		printf("  Warp size: %d\n", prop.warpSize);
  	}
	return 0;
}
