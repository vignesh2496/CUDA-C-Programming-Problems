
#include <hip/hip_runtime.h>
#include<iostream>
#define THREADS_PER_BLOCK 256
#define BLOCKS 128
#define N (1 << 16)

using namespace std;

__global__ void add_array(float A[], float blocks[])
{
	__shared__ int array_per_block[THREADS_PER_BLOCK];
        int global_thread_ID = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x, step = gridDim.x * THREADS_PER_BLOCK, my_sum = 0, num_threads = THREADS_PER_BLOCK;
        for(int i = global_thread_ID; i < N; i += step)
                my_sum += A[i];
	array_per_block[threadIdx.x] = my_sum;
	__syncthreads();
	while(threadIdx.x < num_threads && num_threads > 1) 
        {
		if(threadIdx.x < num_threads / 2)
			array_per_block[threadIdx.x] +=  array_per_block[threadIdx.x + num_threads / 2];
		num_threads = num_threads >> 1;
		__syncthreads();
	}
	if(threadIdx.x == 0)
		blocks[blockIdx.x] = array_per_block[0];
}

void init_array(float A[])
{
        for(int i = 0; i < N; i++)
                A[i] = 1;
}

int main()
{
        float *host_A = new float[N], *host_blocks = new float[BLOCKS], *cuda_A, *cuda_blocks, final_sum = 0;
        init_array(host_A);
        hipMalloc(&cuda_A, sizeof(float) * N);
	hipMemcpy(cuda_A, host_A, sizeof(float) * N, hipMemcpyHostToDevice);
        hipMalloc(&cuda_blocks, sizeof(float) * BLOCKS);
	add_array<<<BLOCKS, THREADS_PER_BLOCK>>>(cuda_A, cuda_blocks);
	hipMemcpy(host_blocks, cuda_blocks, sizeof(float) * BLOCKS, hipMemcpyDeviceToHost); 
	for(int i = 0; i < BLOCKS; i++)
		final_sum += host_blocks[i];
	cout << "Final Sum : " << final_sum << endl;
	free(host_A); free(host_blocks);
	hipFree(cuda_A); hipFree(cuda_blocks);
        return 0;
}

