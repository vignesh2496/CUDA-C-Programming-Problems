
#include <hip/hip_runtime.h>
#include<iostream>
#define M 6
#define N 6
#define THREADS_PER_BLOCK 256

using namespace std;

__global__ void add_matrix(int mat_1[], int mat_2[], int mat_sum[])
{
	int global_thread_ID = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	if(global_thread_ID < M * N)
		mat_sum[global_thread_ID] = mat_1[global_thread_ID] + mat_2[global_thread_ID];
}

void print_matrix(int mat[])
{
	for(int i = 0; i < M; i++)
	{
		for(int j = 0; j < N; j++)
			cout << mat[i * N + j] << "  ";
		cout << endl;
	}
	cout << endl;
} 

void init_matrix(int mat[])
{
	for(int i = 0; i < M; i++)
		for(int j = 0; j < N; j++)
			mat[i * N + j] = i * N + j; 
}

int main()
{
	int *host_A = new int[M * N], *host_B = new int[M * N], *host_sum = new int[M * N], *cuda_A, *cuda_B, *cuda_sum;
	int blocks = M * N / THREADS_PER_BLOCK + ((M * N % THREADS_PER_BLOCK == 0) ? 0 : 1); 
	init_matrix(host_A);
	cout << "A:\n"; 
	print_matrix(host_A);
	init_matrix(host_B);
	cout << "B:\n";
	print_matrix(host_B);
	hipMalloc(&cuda_A, sizeof(int) * M * N);
	hipMalloc(&cuda_B, sizeof(int) * M * N);
	hipMalloc(&cuda_sum, sizeof(int) * M * N);
	hipMemcpy(cuda_A, host_A, sizeof(int) * M * N, hipMemcpyHostToDevice);
	hipMemcpy(cuda_B, host_B, sizeof(int) * M * N, hipMemcpyHostToDevice);
	add_matrix<<<blocks, THREADS_PER_BLOCK>>>(cuda_A, cuda_B, cuda_sum);
	hipMemcpy(host_sum, cuda_sum, sizeof(int) * M * N, hipMemcpyDeviceToHost);
	cout << "A + B:\n";
	print_matrix(host_sum);
	free(host_A); free(host_B); free(host_sum);
	hipFree(cuda_A); hipFree(cuda_B); hipFree(cuda_sum);
	return 0;
}
