#include "hip/hip_runtime.h"
#include <bits/stdc++.h>

#define N (1<<20)
#define NUM_BINS 4096

using namespace std;


void init_array( unsigned int a[], int k, int size)
{
        for( int i = 0; i < size; i++)
       	{
		if(k == 0)
        	a[i] = rand() % (NUM_BINS);
		else if(k == 1)
		a[i] = 0;
      	}
}

__global__ void hist(unsigned int cuda_input[],unsigned int cuda_result[])
{
        __shared__ unsigned int histogram[NUM_BINS];
        int ID = threadIdx.x + blockIdx.x * blockDim.x;
	for( int i = threadIdx.x ; i < NUM_BINS ; i+= (blockDim.x))
	{
		histogram[i] = 0;
	}

	__syncthreads();
        for(int  i = ID ; i < N; i += (gridDim.x * blockDim.x))
        {
                atomicAdd(&(histogram[cuda_input[i]]),1);
        }
        __syncthreads();

        for( int i = threadIdx.x ; i < NUM_BINS ; i += blockDim.x)
        {
                atomicAdd(&(cuda_result[i]), histogram[i]);
        }
}

int main()
{
        unsigned int * input = (unsigned int *)malloc(sizeof(unsigned int) * N) ;
        unsigned int * result = (unsigned int *)malloc(sizeof(unsigned int) * NUM_BINS) ;
        init_array(input, 0, N);
        unsigned int *cuda_input, *cuda_result;
        hipMalloc(&cuda_input,sizeof(unsigned int) * N);
        hipMalloc(&cuda_result,sizeof(unsigned int) * NUM_BINS);
        init_array(result, 1, NUM_BINS);
	hipMemcpy(cuda_input, input, sizeof(unsigned int) * N, hipMemcpyHostToDevice);
        hipMemcpy(cuda_result, result, sizeof(unsigned int) * NUM_BINS, hipMemcpyHostToDevice);
        int block_dim = 256;
        int grid_dim = 256;
        hist <<< block_dim, grid_dim >>> (cuda_input,cuda_result); 
        hipMemcpy(result, cuda_result, sizeof(unsigned int) * NUM_BINS , hipMemcpyDeviceToHost);
        for(int i = 0 ; i < NUM_BINS ; i++)
        {	
		result[i] = min(result[i], 127);
                printf("%u ",result[i]);
        }
        free(input); 
        free(result);
        hipFree(cuda_input); 
        hipFree(cuda_result);
        return 0;
}

