
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#define N 16

using namespace std;

__global__ void RGBtoGray(float img[], float gray_img[])
{
        int ID = threadIdx.x + blockIdx.x * blockDim.x;
        for(int  i = ID ; i < N * N; i += gridDim.x * blockIdx.x)
        {
                gray_img[i] = 0.21 * img[i * 3] + 0.71 * img[i * 3 + 1] + 0.07 * img[i * 3 + 2];
        }
        __syncthreads();

}

void print_matrix(float mat[])
{
        for(int k = 0; k < 3; k++)
        {
                for(int i = 0; i < N; i++)
                {
                        for(int j = 0; j < N; j++)
                                cout << mat[(i * N + j) * 3 + k] << "  ";
                        cout << endl;
                }
                cout << endl << endl;
        }
}

void print_gray_matrix(float mat[])
{
        for(int i = 0; i < N; i++)
        {
                for(int j = 0; j < N; j++)
                        cout << mat[(i * N + j)] << "  ";
                cout << endl;
        }
        cout << endl << endl;
}

void init_matrix(float mat[])
{
        for(int k = 0; k < 3; k++)
        {
                for(int i = 0; i < N; i++)
                        for(int j = 0; j < N; j++)
                                mat[(i * N + j) * 3 + k] = k + 1;
        }
}


int main()
{
        float *host_img = new float[N * N * 3], *host_gray_img = new float[N * N ], *cuda_img, *cuda_gray_img;
        // Assuming N is a multiple of 16 
        //dim3 grid_dim(N / 16, N / 16), block_dim(16, 16);
        int block_dim = 256, grid_dim;
        if( (N * N) % 256 == 0)
        {
                grid_dim = (N * N) / 256;
        }
        else
        {
               	grid_dim = (N * N) / 256 + 1;
        }
        init_matrix(host_img);
        print_matrix(host_img);
        hipMalloc(&cuda_img, sizeof(float) * N * N * 3);
        hipMalloc(&cuda_gray_img, sizeof(float) * N * N );
        hipMemcpy(cuda_img, host_img, sizeof(float) * N * N * 3, hipMemcpyHostToDevice);
        RGBtoGray<<<grid_dim, block_dim>>>(cuda_img, cuda_gray_img);
        hipMemcpy(host_gray_img, cuda_gray_img, sizeof(float) * N * N , hipMemcpyDeviceToHost);
        print_gray_matrix(host_gray_img);
        free(host_img); 
        free(host_gray_img);
        hipFree(cuda_img); 
        hipFree(cuda_gray_img);
        return 0;
}

