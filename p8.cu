
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#define N 16
#define K_SIZE 3
#define BLOCK_DIM 16

using namespace std;

__global__ void convolve(float img[], float kernel[], float conv_img[])
{
	int global_x = threadIdx.x + blockIdx.x * blockDim.x, global_y = threadIdx.y + blockIdx.y * blockDim.y, global_ID = N * global_y + global_x;
	int conv_size = blockDim.x + K_SIZE - 1, pad = K_SIZE / 2;
	__shared__ float block_sub_matrix[BLOCK_DIM + K_SIZE - 1][BLOCK_DIM + K_SIZE - 1][3];

	for(int k = 0; k < 3; k++)
	{
		// Left-Top
		if(global_y - pad >= 0 && global_x - pad >= 0)
			block_sub_matrix[threadIdx.y][threadIdx.x][k] = img[((global_y - pad) * N + (global_x - pad)) * 3 + k];
		else
			block_sub_matrix[threadIdx.y][threadIdx.x][k] = 0;

		// Right-Top
		if(global_y - pad >= 0 && global_x + pad < N)
			block_sub_matrix[threadIdx.y][threadIdx.x + K_SIZE - 1][k] = img[((global_y - pad) * N + (global_x + pad)) * 3 + k];
		else
			block_sub_matrix[threadIdx.y][threadIdx.x + K_SIZE - 1][k] = 0;

		// Left-Bottom
		if(global_y + pad < N && global_x - pad >= 0)
			block_sub_matrix[threadIdx.y + K_SIZE - 1][threadIdx.x][k] = img[((global_y + pad) * N + (global_x - pad)) * 3 + k];
		else
			block_sub_matrix[threadIdx.y + K_SIZE - 1][threadIdx.x][k] = 0;

		// Right-Bottom
		if(global_y + pad < N && global_x + pad < N)
			block_sub_matrix[threadIdx.y + K_SIZE - 1][threadIdx.x + K_SIZE - 1][k] = img[((global_y + pad) * N + (global_x + pad)) * 3 + k];
		else
			block_sub_matrix[threadIdx.y + K_SIZE - 1][threadIdx.x + K_SIZE - 1][k] = 0;
	}

	__syncthreads();

	for(int k = 0; k < 3; k++)
	{
		conv_img[global_ID * 3 + k] = 0;
		for(int y = 0; y < K_SIZE; y++)
			for(int x = 0; x < K_SIZE; x++)
				conv_img[global_ID * 3 + k] += block_sub_matrix[threadIdx.y + K_SIZE - 1 - y][threadIdx.x + K_SIZE - 1 - x][k] * kernel[y * K_SIZE + x];
	}
}

void print_matrix(float mat[])
{
	for(int k = 0; k < 3; k++)
	{
		for(int i = 0; i < N; i++)
		{
			for(int j = 0; j < N; j++)
				cout << mat[(i * N + j) * 3 + k] << "  ";
			cout << endl;
		}
		cout << endl << endl;
	}
} 

void init_matrix(float mat[])
{
	for(int k = 0; k < 3; k++)
	{
		for(int i = 0; i < N; i++)
			for(int j = 0; j < N; j++)
				mat[(i * N + j) * 3 + k] = 1;
	} 
}

void init_kernel(float kernel[])
{
	for(int i = 0; i < K_SIZE; i++)
		for(int j = 0; j < K_SIZE; j++)
			kernel[i * K_SIZE + j] = 1.0 / 9;
  
}

int main()
{
	float *host_img = new float[N * N * 3], *host_kernel = new float[K_SIZE * K_SIZE], *host_conv_img = new float[N * N * 3], *cuda_img, *cuda_kernel, *cuda_conv_img;
	// Assuming N is a multiple of 16 
	dim3 grid_dim(N / BLOCK_DIM, N / BLOCK_DIM), block_dim(BLOCK_DIM, BLOCK_DIM);
	init_matrix(host_img);
	print_matrix(host_img);
	init_kernel(host_kernel); 
	hipMalloc(&cuda_img, sizeof(float) * N * N * 3);
	hipMalloc(&cuda_kernel, sizeof(float) * K_SIZE * K_SIZE);
	hipMalloc(&cuda_conv_img, sizeof(float) * N * N * 3);
	hipMemcpy(cuda_img, host_img, sizeof(float) * N * N * 3, hipMemcpyHostToDevice);
	hipMemcpy(cuda_kernel, host_kernel, sizeof(float) * K_SIZE * K_SIZE, hipMemcpyHostToDevice);
	convolve<<<grid_dim, block_dim>>>(cuda_img, cuda_kernel, cuda_conv_img);
	hipMemcpy(host_conv_img, cuda_conv_img, sizeof(float) * N * N * 3, hipMemcpyDeviceToHost);
    	print_matrix(host_conv_img);
	free(host_img); free(host_kernel); free(host_conv_img);
	hipFree(cuda_img); hipFree(cuda_kernel); hipFree(cuda_conv_img);
	return 0;
}
